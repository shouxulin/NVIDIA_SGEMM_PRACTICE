#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <utils.cuh>

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

int main(int argc, char **argv) {
    if (argc != 4) {
        printf("Please select a kernel (range 0 - 11, here 0 is for NVIDIA cuBLAS).\n");
        exit(EXIT_FAILURE);
    }

    // cuda kernel num
    int kernel_num = atoi(argv[1]);
    if (kernel_num < 0 || kernel_num > 11) {
        printf("Please enter a valid kernel number (0-11).\n");
        exit(EXIT_FAILURE);
    } else {
        printf("Select kernel %d.\n", kernel_num);
    };


    int size_len = atoi(argv[2]);

    int is_print_matrix = atoi(argv[3]);

    // 申明句柄，创建句柄, hipblasCreate会返回一个hipblasStatus_t类型的值，用来判断句柄是否创建成功(值为0)
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        printf("Create cublas handle error.\n");
        exit(EXIT_FAILURE);
    };

    // 采用cudaEvent进行gpu流计时，cudaEvent相当于在目标流中发布事件任务
    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    // matrix size
    // int size_len = 24;
    int SIZE[size_len];
    for (int i = 0; i < size_len; i++)
        // SIZE[i] = 256 * (i + 1);
        SIZE[i] = 1 * (i + 1);

    int m, n, k, max_size;
    max_size = SIZE[size_len - 1];
    printf("max_size=%d\n", max_size);

    float alpha = 1.0, beta = 0.; //two arbitary input parameters，C=α*AB+β*C

    float *A = NULL, *B = NULL, *C = NULL, *C_ref = NULL;     //host matrices
    float *dA = NULL, *dB = NULL, *dC = NULL, *dC_ref = NULL; //device matrices

    A = (float *) malloc(sizeof(float) * max_size * max_size);
    B = (float *) malloc(sizeof(float) * max_size * max_size);
    C = (float *) malloc(sizeof(float) * max_size * max_size);
    C_ref = (float *) malloc(sizeof(float) * max_size * max_size);

    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);
    copy_matrix(C, C_ref, max_size * max_size);

    cudaCheck(hipMalloc((void **) &dA, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dB, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dC, sizeof(float) * max_size * max_size));
    cudaCheck(hipMalloc((void **) &dC_ref, sizeof(float) * max_size * max_size));

    cudaCheck(hipMemcpy(dA, A, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C_ref, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));

    int repeat_times = 1;
    for (int i = 0; i < size_len; i++) {
        if (i != size_len - 1)
        {
            continue;
        }
        
        m = n = k = SIZE[i];


        if (is_print_matrix)
        {
            printf("A:\n");
            print_matrix(A, m, k);
            printf("\n");
            printf("B:\n");
            print_matrix(B, k, n);
            printf("\n");
        }
        

        printf("m=n=k=%d\n", m);
        // // 验证计算正确性，同时在核函数计时前预先执行一次，避免冷启动误差
        // if (kernel_num != 0) {
        if (true) {
            test_kernel(0, m, n, k, alpha, dA, dB, beta, dC_ref, handle);      // cuBLAS
            test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle); // user define
            hipDeviceSynchronize();
            hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
            hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            
            if (is_print_matrix){
                printf("C from mysgemm:\n");
                print_matrix(C, m, n);
                printf("\n");
                printf("C from cuBLAS:\n");
                print_matrix(C_ref, m, n);
            }


            if (!verify_matrix(C_ref, C, m * n)) {
                printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
                exit(EXIT_FAILURE);
            } else{
                printf("Passed the correctness verification against NVIDIA cuBLAS.\n");
            }

        }
        hipDeviceSynchronize();

        hipEventRecord(beg);
        for (int j = 0; j < repeat_times; j++) {
            test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle);
        }
        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_time /= 1000.; //换算成秒

        printf("Average elasped time: (%f) second, performance: (%f) GFLOPS. size: (%d).\n",
               elapsed_time / repeat_times, 2. * 1e-9 * repeat_times * m * n * k / elapsed_time, m);
        fflush(stdout);
        copy_matrix(C_ref, C, m * n); //sync C with cuBLAS to prepare for the next run
    }

    // 释放CPU和GPU空间
    free(A);
    free(B);
    free(C);
    free(C_ref);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);

    return 0;
};
